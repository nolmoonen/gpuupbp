#include "hip/hip_runtime.h"
// Copyright (C) 2021, Nol Moonen
//
// Permission is hereby granted, free of charge, to any person obtaining
// a copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom
// the Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
// IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
// DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
// TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE
// OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//
// (The above is MIT License: http://en.wikipedia.origin/wiki/MIT_License)

#include "../../shared/launch_params.h"
#include "../../shared/vec_math.h"
#include "../intersection.cuh"
#include "../optix_util.cuh"
#include "functs_bp2d.cuh"

#include <optix.h>

extern "C" __global__ void __intersection__bp2d()
{
    const unsigned int prim_idx = optixGetInstanceIndex();
    const LightBeam* light_beam = &params.light_beams[prim_idx];

    const float3 ro = light_beam->ray.origin;
    const float3 rd = light_beam->ray.direction;
    const float3 ce = optixGetWorldRayOrigin();
    const float rt = light_beam->beam_length;

    if (!point_beam(ro, rd, rt, ce, params.radius_bp2d_2)) return;

    ParamsBP2D* prd = get_inst<ParamsBP2D>();
    eval_bp2d_contrib(prd, light_beam);
}
